
#include "hip/hip_complex.h"
#include "hermitian_kernel.h"
#include "hresult.h"
#include "kernel_helpers.h"
#include <chrono>
#include <complex> // Include the necessary header file
#include <cstring> // For std::memcpy
#include <hip/hip_runtime.h>
#include <iostream>
#include <string_view>
#include <vector>

using namespace plinalg;

int main(int argc, char *argv[]) {
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Example descriptor
  HermitianDescriptor descriptor = {10, 5,
                                    5}; // Batch size of 1, matrix size of 4

  // Pack the descriptor
  auto packedDescriptor = PackDescriptorAsString(descriptor);

  // Allocate and initialize input and output buffers
  const int matrix_size = descriptor.m_rows * descriptor.n_cols;
  const int global_size = matrix_size * descriptor.batch_size;
  std::vector<hipComplex> hostInput(global_size);

  float real_start = 0.5f * matrix_size;
  float imag_start = -0.5f * matrix_size;

  for (int j = 1; j <= descriptor.batch_size; j++)
    for (int i = 0; i < matrix_size; ++i) {
      float real = real_start - i;
      float imag = imag_start + i;
      hostInput[i * j] = make_hipComplex(real, imag);
    }

  auto start = std::chrono::high_resolution_clock::now();

  std::vector<hipComplex> hostExpected(global_size);

  for (int j = 1; j <= descriptor.batch_size; j++)
    for (int i = 0; i < descriptor.m_rows; ++i) {
      for (int k = 0; k < descriptor.n_cols; ++k) {
        hipComplex element =
            hostInput[i * descriptor.n_cols + k + (j - 1) * matrix_size];
        hostExpected[k * descriptor.m_rows + i + (j - 1) * matrix_size] =
            make_hipComplex(hipCrealf(element), -hipCimagf(element));
      }
    }

  hipDeviceSynchronize();
  // Stop timer
  auto stop = std::chrono::high_resolution_clock::now();

  // Calculate duration
  auto duration =
      std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

  std::cout << "Time taken by cpu hermitian: " << duration.count()
            << " microseconds" << std::endl;

  std::vector<hipComplex> hostOutput(global_size);

  hipComplex *deviceInput, *deviceOutput;
  hipMalloc(&deviceInput, global_size * sizeof(hipComplex));
  hipMalloc(&deviceOutput, global_size * sizeof(hipComplex));

  hipMemcpy(deviceInput, hostInput.data(), global_size * sizeof(hipComplex),
             hipMemcpyHostToDevice);

  void *buffers[2] = {deviceInput, deviceOutput};

  // Call the Hermitian function
  start = std::chrono::high_resolution_clock::now();
  HRESULT result = Hermitian(stream, buffers, packedDescriptor.c_str(),
                             packedDescriptor.size());
  if (result != S_OK) {
    std::cerr << "Hermitian kernel launch failed." << std::endl;
    return -1;
  }
  hipDeviceSynchronize();
  // Stop timer
  stop = std::chrono::high_resolution_clock::now();

  // Calculate duration
  duration =
      std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

  std::cout << "Time taken by cuda kernel: " << duration.count()
            << " microseconds" << std::endl;
  // Retrieve and print the result
  hipMemcpy(hostOutput.data(), deviceOutput, global_size * sizeof(hipComplex),
             hipMemcpyDeviceToHost);

  if (false) {
      for (int batch = 0; batch < descriptor.batch_size; ++batch) {
        std::cout << "**************************" << std::endl;
        std::cout << "Starting Batch[" << batch << "]" << std::endl;
        std::cout << "**************************" << std::endl;
        for (int row = 0; row < descriptor.m_rows; ++row) {
          for (int col = 0; col < descriptor.n_cols; ++col) {
            int index = batch * descriptor.m_rows * descriptor.n_cols +
                        row * descriptor.n_cols + col;
            std::cout << "Batch[" << batch << "] Row[" << row << "] Col[" << col
                      << "] = (Re = " << hipCrealf(hostOutput[index])
                      << ", Img = " << hipCimagf(hostOutput[index]) << "i), "
                      << "Expected = (Re = " << hipCrealf(hostExpected[index])
                      << ", Img = " << hipCimagf(hostExpected[index]) << "i), "
                      << "Original Input = (Re = " << hipCrealf(hostInput[index])
                      << ", Img = " << hipCimagf(hostInput[index]) << "i)"
                      << std::endl;
          }
        }
      }
  }

  // Cleanup
  hipFree(deviceInput);
  hipFree(deviceOutput);
  hipStreamDestroy(stream);

  return 0;
}